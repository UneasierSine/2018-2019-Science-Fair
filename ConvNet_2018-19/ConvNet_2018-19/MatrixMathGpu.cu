#include "hip/hip_runtime.h"
#include <iostream>
#include "MatrixMath.h"
#include "MatrixMath.cuh"

using namespace thrust;

struct add
{
	add() {}
	__host__ __device__ double operator()(double a, double b)
	{
		return a + b;
	}
};

struct sub
{
	sub() {}
	__host__ __device__ double operator()(double a, double b)
	{
		return a - b;
	}
};

struct mul
{
	mul() {}
	__host__ __device__ double operator() (double a, double b)
	{
		return a * b;
	}
};

struct div
{
	div() {}
	__host__ __device__ double operator() (double a, double b)
	{
		return a / b;
	}
};

struct pow
{
	pow() {}
	__host__ __device__ double operator() (double a, double b)
	{
		return std::pow(a, b);
	}
};

struct rad
{
	rad() {}
	__host__ __device__ double operator() (double a, double b)
	{
		return pow(a, 1/b);
	}
};

vector<double> addMatTermsGpu(vector<double> vec1, vector<double> vec2)
{
	vector<double> returnVec(1);

	if (vec1.size() != vec2.size())
	{
		returnVec[0] = -1;
		return returnVec;
	}

	host_vector<double> h_v1 = vec1;
	host_vector<double> h_v2 = vec2;

	device_vector<double> d_rV(vec1.size());

	device_vector<double> d_v1 = h_v1;
	device_vector<double> d_v2 = h_v1;

	transform(d_v1.begin(), d_v1.end(), d_v2.begin, d_rV.begin(), add());

	for (int i = 0; i < d_rV.size(); i++)
	{
		returnVec[i] = d_rV[i];
	}

	return returnVec;
}

vector<double> subMatTermsGpu(vector<double> vec1, vector<double> vec2)
{

}

vector<double> mulMatTermsGpu(vector<double> vec1, vector<double> vec2)
{

}

vector<double> divMatTermsGpu(vector<double> vec1, vector<double> vec2)
{

}

vector<double> powMatTermsGpu(vector<double> vec1, vector<double> vec2)
{

}

vector<double> radMatTermsGpu(vector<double> vec1, vector<double> vec2)
{

}

double sumTermsGpu(vector<double> vector)
{

}

double dotProductGpu(vector<double> vec1, vector<double> vec2)
{

}